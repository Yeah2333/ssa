// g2o - General Graph Optimization
// Copyright (C) 2011 M. Ruhnke, 
// 
// g2o is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published
// by the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// g2o is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
// 
// You should have received a copy of the GNU Lesser General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include "pcg_cusp.h"

#include <cusp/precond/diagonal.h>
#include <cusp/transpose.h>
#include <cusp/elementwise.h>
#include <cusp/csr_matrix.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/bicgstab.h>

#include "EXTERNAL/g2o/g2o/stuff/timeutil.h"

namespace g2o {

    SolverCUSPCG::SolverCUSPCG() {};

    SolverCUSPCG::~SolverCUSPCG() {};

    bool SolverCUSPCG::solve(double *A, int *ptr, int *indices, int cols, int nz, double *x, double *b) {

        cusp::array1d<double, cusp::host_memory> x_cusp_host;
        cusp::csr_matrix<int, double, cusp::device_memory> A_ccs_cusp_device(cols, cols, nz);
        cusp::array1d<double, cusp::device_memory> x_cusp_device(A_ccs_cusp_device.num_rows, 0.0);
        cusp::array1d<double, cusp::device_memory> b_cusp_device(A_ccs_cusp_device.num_rows);

        thrust::copy(A, A + nz, A_ccs_cusp_device.values.begin());
        thrust::copy(indices, indices + nz, A_ccs_cusp_device.column_indices.begin());
        thrust::copy(ptr, ptr + cols + 1, A_ccs_cusp_device.row_offsets.begin());
        thrust::copy(b, b + cols, b_cusp_device.begin());

//     cusp::csr_matrix<int,double,cusp::device_memory>  At_ccs_cusp_device(cols, cols, nz);
//     cusp::transpose(A_ccs_cusp_device, At_ccs_cusp_device);
//     cusp::csr_matrix<int,double,cusp::host_memory>  At_ccs_cusp_host;
//   
//     //removing diagonal of transposed matrix
//     At_ccs_cusp_host = At_ccs_cusp_device;
//     int c=0;
//     for(int i=0;i < (At_ccs_cusp_host.row_offsets.size()-1); ++i){
//       int c_start = At_ccs_cusp_host.row_offsets[i];
//       int c_end = At_ccs_cusp_host.row_offsets[i+1];
//       for(int j=c_start;j < c_end; ++j){
//         if(At_ccs_cusp_host.column_indices[j] == c)
//           At_ccs_cusp_host.values[j] = 0.0;
//       }
//       c++;
//     }
//     At_ccs_cusp_device = At_ccs_cusp_host;
//     cusp::add(A_ccs_cusp_device, At_ccs_cusp_device, A_ccs_cusp_device);

        cusp::convergence_monitor<double> monitor(b_cusp_device, cols, 1e-8);

        // set preconditioner (identity)
        //cusp::identity_operator<double,cusp::device_memory> M(A_ccs_cusp_device.num_rows, A_ccs_cusp_device.num_rows);

        // setup preconditioner Jacoby
        cusp::precond::diagonal<double, cusp::device_memory> M(A_ccs_cusp_device);

        // solve the linear system A * x = b with the Conjugate Gradient method
        cusp::krylov::cg(A_ccs_cusp_device, x_cusp_device, b_cusp_device, monitor, M);
        x_cusp_host = x_cusp_device;
        // copy x back
        for (int i = 0; i < cols; i++) {
            x[i] = x_cusp_host[i];
        }
        return true;
    }

    bool SolverCUSPCG::solveWithTiming(double *A, int *ptr, int *indices, int cols, int nz, double *x, double *b) {

        double timing = get_time();
        cerr << "allocating memory...  \t ";
        cusp::array1d<double, cusp::host_memory> x_cusp_host;

        // copy to the device
        cusp::csr_matrix<int, double, cusp::device_memory> A_ccs_cusp_device(cols, cols, nz);
        cusp::array1d<double, cusp::device_memory> x_cusp_device(A_ccs_cusp_device.num_rows, 0.0);
        cusp::array1d<double, cusp::device_memory> b_cusp_device(A_ccs_cusp_device.num_rows);
        cerr << "done in " << (get_time() - timing) * 1000 << " ms." << endl;

        timing = get_time();
        cerr << "copy data to gpu...  \t";
        thrust::copy(A, A + nz, A_ccs_cusp_device.values.begin());
        thrust::copy(indices, indices + nz, A_ccs_cusp_device.column_indices.begin());
        thrust::copy(ptr, ptr + cols + 1, A_ccs_cusp_device.row_offsets.begin());
        thrust::copy(b, b + cols, b_cusp_device.begin());
        cerr << "done in " << (get_time() - timing) * 1000 << " ms." << endl;

//     timing = get_time();
//     cerr << "csr upper to csr full ... \t";
//     cusp::csr_matrix<int,double,cusp::device_memory>  At_ccs_cusp_device(cols, cols, nz);
//     //removing diagonal of transposed matrix
//     //At_ccs_cusp_host = At_ccs_cusp_device;
//     int col = 0;
//     for(int i=0;i < cols; ++i){
//       int c_start = ptr[i];
//       int c_end = ptr[i+1];
//       for(int j=c_start;j < c_end; ++j){
//         int row = indices[j];
//         if(col == row)
//           A[j] = 0.0;
//       }
//       col++;
//     }
//     thrust::copy(A, A + nz, At_ccs_cusp_device.values.begin());
//     At_ccs_cusp_device.column_indices = A_ccs_cusp_device.column_indices;
//     At_ccs_cusp_device.row_offsets = A_ccs_cusp_device.row_offsets;
// 
//     cusp::csr_matrix<int,double,cusp::device_memory>  Atmp(cols, cols, nz);
//     cusp::transpose(At_ccs_cusp_device, Atmp);
//     cusp::add(A_ccs_cusp_device, Atmp, A_ccs_cusp_device);
//     cerr << "done in " << (get_time() - timing) * 1000 << " ms." << endl;

        cerr << "running conjugate gradient with jacoby preconditioner... \t";
        // set stopping criteria:
        //  iteration_limit    = 100
        //  relative_tolerance = 1e-3
        cusp::convergence_monitor<double> monitor(b_cusp_device, cols, 1e-8);

        // set preconditioner (identity)
        //cusp::identity_operator<double,cusp::device_memory> M(A_ccs_cusp_device.num_rows, A_ccs_cusp_device.num_rows);

        // setup preconditioner Jacoby
        cusp::precond::diagonal<double, cusp::device_memory> M(A_ccs_cusp_device);

        // solve the linear system A * x = b with the Conjugate Gradient method
        cusp::krylov::cg(A_ccs_cusp_device, x_cusp_device, b_cusp_device, monitor, M);
        cerr << "done in " << (get_time() - timing) * 1000 << " ms." << endl;

        timing = get_time();
        cerr << "copy result back to host memory... \t";
        x_cusp_host = x_cusp_device;
        // copy x back
        for (int i = 0; i < cols; i++) {
            x[i] = x_cusp_host[i];
        }
        cerr << "done in " << (get_time() - timing) * 1000 << " ms." << endl;
        return true;
    }

}// end namespace


