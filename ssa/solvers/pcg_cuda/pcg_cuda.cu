#include "hip/hip_runtime.h"
// g2o - General Graph Optimization
// Copyright (C) 2011 M. Ruhnke, 
// 
// g2o is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published
// by the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// g2o is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
// 
// You should have received a copy of the GNU Lesser General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#include "pcg_cuda.h"
// Utilities and system includes
#include <shrUtils.h>
#include <shrQATest.h>
#include <cutil_inline.h>
#include <hipsparse.h>
#include <hipblas.h>

namespace g2o {

    SolverCUDACG::SolverCUDACG() {};

    SolverCUDACG::~SolverCUDACG() {};

    bool SolverCUDACG::solve(double *A, int *ptr, int *indices, int cols, int nz, double *x, double *b) {
        hipsparseHandle_t handle;
        hipsparseStatus_t status;
        hipsparseMatDescr_t descr;

        /* Get handle to the CUSPARSE context */
        status = hipsparseCreate(&handle);
        /* Description of the A matrix*/
        status = hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_UPPER);

        if (status != 0)
            return false;
        /** copy ccs to gpu */
        //Sparse matrix
        double *cu_A_ccs;
        int *cu_col_ptr_ccs, *cu_row_idx_ccs;

        double *cu_x, *cu_b;
        double *cu_p, *cu_Ap;
        double *cu_r;

        //alloc memory
        cutilSafeCall(hipMalloc((void **) &cu_col_ptr_ccs, sizeof(int) * (cols + 1)));
        cutilSafeCall(hipMalloc((void **) &cu_row_idx_ccs, sizeof(int) * nz));
        cutilSafeCall(hipMalloc((void **) &cu_A_ccs, sizeof(double) * nz));
        cutilSafeCall(hipMalloc((void **) &cu_x, sizeof(double) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_b, sizeof(double) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_r, sizeof(double) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_p, sizeof(double) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_Ap, sizeof(double) * cols));

        hipMemcpy(cu_A_ccs, A, sizeof(double) * nz, hipMemcpyHostToDevice);
        hipMemcpy(cu_row_idx_ccs, indices, sizeof(int) * nz, hipMemcpyHostToDevice);
        hipMemcpy(cu_col_ptr_ccs, ptr, sizeof(int) * (cols + 1), hipMemcpyHostToDevice);
        hipMemcpy(cu_x, x, sizeof(double) * cols, hipMemcpyHostToDevice);
        hipMemcpy(cu_b, b, sizeof(double) * cols, hipMemcpyHostToDevice);

        /** Solve CG problem */
        int max_iter = cols;
        int iteration = 0;
        double tol = 1e-6;
        double alpha, beta;

        /**  r=b-A*x; */
        /** r = b; */
        hipblasDcopy(cols, cu_b, 1, cu_r, 1);
        /** r = -1.0 * A * x + 1.0 * r; (r = b previous step) */
        hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, -1.0,
                       descr, cu_A_ccs, cu_col_ptr_ccs, cu_row_idx_ccs, cu_x, 1.0, cu_r);
        /** p = r; */
        hipblasDcopy(cols, cu_r, 1, cu_p, 1);
        /** rs_old = r'*r; */
        double rs_old = hipblasDdot(cols, cu_r, 1, cu_r, 1);
        double rs_new = 0;

        while (rs_old > tol * tol && iteration <= max_iter) {
            iteration++;

            hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, 1.0,
                           descr, cu_A_ccs, cu_col_ptr_ccs, cu_row_idx_ccs, cu_p, 0.0,
                           cu_Ap);   /** Ap = 1.0 * A * p + 0.0 * y; */
            alpha = rs_old / hipblasDdot(cols, cu_Ap, 1, cu_p, 1);  /** alpha=rs_old/(p'*Ap); */
            hipblasDaxpy(cols, alpha, cu_p, 1, cu_x, 1);          /** x=x+alpha*p; */
            hipblasDaxpy(cols, -alpha, cu_Ap, 1, cu_r, 1);        /** r=r-alpha*Ap; */
            rs_new = hipblasDdot(cols, cu_r, 1, cu_r, 1);          /** rsnew=r'*r; */
            /** p=r+rsnew/rsold*p; */
            beta = rs_new / rs_old; /** beta = rsnew/rsold */
            hipblasDscal(cols, beta, cu_p, 1); /** p=beta*p; */
            hipblasDaxpy(cols, 1.0, cu_r, 1, cu_p, 1);/** p=r+p; */
            rs_old = rs_new;
        }
        shrLog("  iteration = %3d, residual = %e \n", iteration, sqrt(rs_old));
        hipMemcpy(x, cu_x, sizeof(double) * cols, hipMemcpyDeviceToHost);

        hipFree(cu_col_ptr_ccs);
        hipFree(cu_row_idx_ccs);
        hipFree(cu_A_ccs);
        hipFree(cu_x);
        hipFree(cu_r);
        hipFree(cu_p);
        hipFree(cu_Ap);
        hipFree(cu_b);
        return true;
    }

    bool SolverCUDACG::solve(float *A, int *ptr, int *indices, int cols, int nz, float *x, float *b) {
        hipsparseHandle_t handle;
        hipsparseStatus_t status;
        hipsparseMatDescr_t descr;

        /* Get handle to the CUSPARSE context */
        status = hipsparseCreate(&handle);
        /* Description of the A matrix*/
        status = hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_UPPER);

        if (status != 0)
            return false;
        /** copy ccs to gpu */
        //Sparse matrix
        float *cu_A_ccs;
        int *cu_col_ptr_ccs, *cu_row_idx_ccs;

        float *cu_x, *cu_b;
        float *cu_p, *cu_Ap;
        float *cu_r;

        //alloc memory
        cutilSafeCall(hipMalloc((void **) &cu_col_ptr_ccs, sizeof(int) * (cols + 1)));
        cutilSafeCall(hipMalloc((void **) &cu_row_idx_ccs, sizeof(int) * nz));
        cutilSafeCall(hipMalloc((void **) &cu_A_ccs, sizeof(float) * nz));
        cutilSafeCall(hipMalloc((void **) &cu_r, sizeof(float) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_x, sizeof(float) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_b, sizeof(float) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_p, sizeof(float) * cols));
        cutilSafeCall(hipMalloc((void **) &cu_Ap, sizeof(float) * cols));

        hipMemcpy(cu_A_ccs, A, sizeof(float) * nz, hipMemcpyHostToDevice);
        hipMemcpy(cu_row_idx_ccs, indices, sizeof(int) * nz, hipMemcpyHostToDevice);
        hipMemcpy(cu_col_ptr_ccs, ptr, sizeof(int) * (cols + 1), hipMemcpyHostToDevice);
        hipMemcpy(cu_x, x, sizeof(float) * cols, hipMemcpyHostToDevice);
        hipMemcpy(cu_b, b, sizeof(float) * cols, hipMemcpyHostToDevice);

        /** Solve CG problem */
        int max_iter = cols;
        int iteration = 0;
        float tol = 1e-8;
        float alpha, beta;

        /**  r=b-A*x; */
        /** r = b; */
        hipblasScopy(cols, cu_b, 1, cu_r, 1);
        /** r = -1.0 * A * x + 1.0 * r; (r = b previous step) */
        hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, -1.0,
                       descr, cu_A_ccs, cu_col_ptr_ccs, cu_row_idx_ccs, cu_x, 1.0, cu_r);
        /** p = r; */
        hipblasScopy(cols, cu_r, 1, cu_p, 1);
        /** rs_old = r'*r; */
        float rs_old = hipblasSdot(cols, cu_r, 1, cu_r, 1);
        float rs_new = 0;

        while (rs_old > tol * tol && iteration <= max_iter) {
            iteration++;

            hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, cols, cols, 1.0,
                           descr, cu_A_ccs, cu_col_ptr_ccs, cu_row_idx_ccs, cu_p, 0.0,
                           cu_Ap);   /** Ap = 1.0 * A * p + 0.0 * y; */
            alpha = rs_old / hipblasSdot(cols, cu_Ap, 1, cu_p, 1);  /** alpha=rs_old/(p'*Ap); */
            hipblasSaxpy(cols, alpha, cu_p, 1, cu_x, 1);          /** x=x+alpha*p; */
            hipblasSaxpy(cols, -alpha, cu_Ap, 1, cu_r, 1);        /** r=r-alpha*Ap; */
            rs_new = hipblasSdot(cols, cu_r, 1, cu_r, 1);          /** rsnew=r'*r; */
            /** p=r+rsnew/rsold*p; */
            beta = rs_new / rs_old; /** beta = rsnew/rsold */
            hipblasSscal(cols, beta, cu_p, 1); /** p=beta*p; */
            hipblasSaxpy(cols, 1.0, cu_r, 1, cu_p, 1);/** p=r+p; */
            rs_old = rs_new;
        }
        shrLog("  iteration = %3d, residual = %e \n", iteration, sqrt(rs_old));
        hipMemcpy(x, cu_x, sizeof(float) * cols, hipMemcpyDeviceToHost);

        hipFree(cu_col_ptr_ccs);
        hipFree(cu_row_idx_ccs);
        hipFree(cu_A_ccs);
        hipFree(cu_x);
        hipFree(cu_r);
        hipFree(cu_p);
        hipFree(cu_Ap);
        hipFree(cu_b);
        return true;
    }

}// end namespace


